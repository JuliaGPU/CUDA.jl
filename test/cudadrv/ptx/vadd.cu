
#include <hip/hip_runtime.h>
extern "C" {

__global__ void vadd(const float *a, const float *b, float *c)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

}
